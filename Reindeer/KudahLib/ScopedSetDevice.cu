#include "ScopedSetDevice.h"

#include "hip/hip_runtime.h"
#include ""

#include "FormatString.hpp"
#include "PlatformSpecific.h"

#include <exception>
#include <stdio.h>

using namespace kudah;

ScopedCUDASetDevice::ScopedCUDASetDevice(unsigned device)
{
	auto const setDeviceStatus = hipSetDevice(static_cast<int>(device));
	if (setDeviceStatus != hipSuccess)
	{
		const auto message = obelisk::formatString("hipSetDevice(%d) failed with code %d", device, setDeviceStatus);
		obelisk::platform_utilities::outputDebugString(message);
		throw std::exception(message.c_str());
	}
}
ScopedCUDASetDevice::~ScopedCUDASetDevice()
{
	if (!doDeviceReset)
		return;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	auto const cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		obelisk::platform_utilities::outputDebugString("hipDeviceReset failed");
	}
}

ScopedCUDASetDevice::ScopedCUDASetDevice(ScopedCUDASetDevice &&o)
{
	// Just prevent the other class's destructor for doing the device reset
	o.doDeviceReset = false;
}
